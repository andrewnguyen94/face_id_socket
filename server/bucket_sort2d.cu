#include "hip/hip_runtime.h"
#include "bucket_sort2d.cuh"

__global__ void bruteforceDistances (float *desQuery, float *desReference , float *distance, int referenceSize , int dim)
{
    //float sdiff[MAX_DATA_DIM];//If put fixed size, then need to care about the sdiff range over tid
    float sdiff, sdiff_sum;

    unsigned int tid = threadIdx.x + blockIdx.x*blockDim.x;
    //printf("Inside KERNEL\n");

    while (tid < referenceSize) {
        sdiff_sum = 0;

        //#pragma unroll: Use to store index, variables in registers, for speeding up
        //But here this does not help since data_dim (also tid, nodeIdx) is not known at compiling time

        int id1 = dim*tid;

#pragma unroll 32
        for (int i=0;i<dim;i++){
            sdiff = desReference[id1+i] - desQuery[i];
            sdiff = sdiff*sdiff;
            sdiff_sum += sdiff;
        }

        distance[tid] = sdiff_sum;

        tid += blockDim.x * gridDim.x;
    }



}

void get_descriptionId_from_db(int *descriptionId, std::vector<PersonInfo> personDatas)
{
	for(size_t i = 0; i < personDatas.size(); ++i){
		descriptionId[i] = (int)i;
	}
}

void get_reference_from_db(float *reference, std::vector<PersonInfo> personDatas)
{
	for(size_t i = 0; i < personDatas.size(); i++){
		dlib::matrix <float, 0, 1> info = personDatas[i].vec128;
		long rows = info.nr();
		long cols = info.nc();
		for(long r = 0; r < info.nr(); ++r){
			for(long c = 0; c < info.nc(); ++c){
				reference[i*rows*cols + r * cols + c] = info(r,c);
			}
		}
	}
}

void get_query_from_db(float *query, std::vector<float> search_vector)
{
	for(size_t i = 0; i < search_vector.size(); ++i){
		query[i] = search_vector[i];
	}
}

std::vector<Pair> get_search_query(std::vector<float> search_vector,
									std::vector<PersonInfo> personDatas, int number_of_vector_request)
{
	std::vector<Pair> result;
	size_t size_of_database = personDatas.size();
/*do something here*/
	int num_dimension = 128;
	int num_reference = size_of_database;
	std::cout << "num_reference :" << num_reference << std::endl;
	int dim = 128;
	
	float *distance = new float[num_reference];
	memset((void*)distance, 0, sizeof(float) * num_reference);
	float *query = new float[num_dimension];
	int *descriptionId = new int[num_reference];
	float *reference = new float[num_dimension * num_reference];
	get_query_from_db(query, search_vector);
	get_descriptionId_from_db(descriptionId, personDatas);
	get_reference_from_db(reference, personDatas);

	float *d_query = NULL;
	CHECK_ERROR(hipMalloc((void**)&d_query, sizeof(float) * num_dimension));
	CHECK_ERROR(hipMemcpy(d_query, query, sizeof(float) * num_dimension,hipMemcpyHostToDevice));

	float *d_reference = NULL;
	CHECK_ERROR(hipMalloc((void**)&d_reference, sizeof(float) * num_dimension * num_reference));
	CHECK_ERROR(hipMemcpy(d_reference, reference, sizeof(float) * num_dimension * num_reference, hipMemcpyHostToDevice));

	int *d_descriptionId = NULL;
	CHECK_ERROR(hipMalloc((void**)&d_descriptionId, sizeof(int) * num_reference));
	CHECK_ERROR(hipMemcpy(d_descriptionId, descriptionId, sizeof(int) * num_reference, hipMemcpyHostToDevice));

	float *d_distance = NULL;
	CHECK_ERROR(hipMalloc((void **)&d_distance, sizeof(float) * num_reference));
	CHECK_ERROR(hipMemcpy(d_distance, distance, sizeof(float) * num_reference, hipMemcpyHostToDevice));

	bruteforceDistances<<< GRID_DIM, BLOCK_DIM >>>(d_query, d_reference, d_distance, num_reference, dim);

	thrust::device_ptr<float> dd_keys(d_distance);
	thrust::device_ptr<int> dd_values(d_descriptionId);

	thrust::sort_by_key(dd_keys, dd_keys + num_reference, dd_values);

	hipMemcpy(distance, d_distance, sizeof(float) * num_reference, hipMemcpyDeviceToHost);
	hipMemcpy(descriptionId, d_descriptionId, sizeof(int) * num_reference, hipMemcpyDeviceToHost);

	if(number_of_vector_request > num_reference)
	{
		number_of_vector_request = num_reference;
	}

	for(int i = 0; i < number_of_vector_request; ++i){
		dlib::matrix <float, 0, 1> info = personDatas[descriptionId[i]].vec128;
		std::vector<float> content;
		for(long r = 0; r < info.nr(); ++r){
			for(long c = 0; c < info.nc(); ++c){
				content.push_back(info(r,c));
			}
		}
		Pair *p = new Pair(personDatas[descriptionId[i]].faceId, content);
		p->set_face_name(personDatas[descriptionId[i]].faceName);
		result.push_back(*p);
	}

	hipFree(d_query);
	hipFree(d_reference);
	hipFree(d_descriptionId);
	hipFree(d_distance);
	delete[] distance;
	delete[] query;
	delete[] descriptionId;
	delete[] reference;

	return result;
}